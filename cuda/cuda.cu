#include "hip/hip_runtime.h"
/*
  Copyright (c) 2017 
  Gregory Gaston, Geoffrey Maggi, Prajyoth Bhandary, and Sriharsha Makineni

  This program is free software: you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation, either version 3 of the License, or
  (at your option) any later version.

  This program is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.

  You should have received a copy of the GNU General Public License
  along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/


#include "../data/readdata.h"
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <sys/time.h>

void getInfo(int *threadsPerBlock, size_t *sharedMemPerBlock) {
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);
  *threadsPerBlock = deviceProp.maxThreadsPerBlock;
  *sharedMemPerBlock = deviceProp.sharedMemPerBlock;
}

__global__
void matrixMultiplyTileKernel(float A[], float B[], float C[], int w) {
  int j = blockDim.x * blockIdx.x + threadIdx.x; //COL
  int i = blockDim.y * blockIdx.y + threadIdx.y; //ROW

  float temp = 0;
  __shared__ float s_A[TILE_WIDTH][TILE_WIDTH];
  __shared__ float s_B[TILE_WIDTH][TILE_WIDTH];


  for(int k = 0; k < w/TILE_WIDTH; k++) {
    s_A[threadIdx.y][threadIdx.x] = A[i * w + k * TILE_WIDTH + threadIdx.x];
    s_B[threadIdx.y][threadIdx.x] = B[(k * TILE_WIDTH + threadIdx.y) * w + j];
    __syncthreads();

    for(int h = 0; h < TILE_WIDTH; h++)
      temp += s_A[threadIdx.y][h] * s_B[h][threadIdx.x];
    __syncthreads();
  }

  C[i * w + j] = temp;
}

void matrixMultiplyCUDA(float A[], float B[], float C[], int n, 
                        int threadPerBlock, size_t sharedMemPerBlock) {
  int size = n * n * sizeof(float);
  float *d_A, *d_B, *d_C;
  
  //Allocate
  hipMalloc((void**) &d_A, size);
  hipMalloc((void**) &d_B, size);
  hipMalloc((void**) &d_C, size);

  //Copy Memory
  hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

  float blk = 32.0;
  dim3 dimGrid(ceil(n/blk), ceil(n/blk), 1);
  dim3 dimBlock(blk, blk, 1);

  //PerformCalculation
  matrixMultiplyTileKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, n);

  //Copy Solution
  hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

  //Free
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
}

int main(int argc, char *argv[]) {
  //Read Device
  int threadPerBlock;
  size_t sharedMemPerBlock;
  getInfo(&threadPerBlock, &sharedMemPerBlock);

  timeval start, stop;

  //Read File(s)
  int m, n;
  char *filename = argv[1];
  float *h_A = readfile(filename, &m, &n);
  float *h_C = (float *) malloc(sizeof(float) * m * n);
  float *h_D = (float *) malloc(sizeof(float) * m * n);

  //Do Computation
  gettimeofday(&start, NULL);
  matrixMultiplyCUDA(h_A, h_A, h_C, n, threadPerBlock, sharedMemPerBlock);
  gettimeofday(&stop, NULL);

  printf("Time to run: %lu microseconds\n", stop.tv_usec - start.tv_usec);

  free(h_A); free(h_C); free(h_D);

 return 0;
}

